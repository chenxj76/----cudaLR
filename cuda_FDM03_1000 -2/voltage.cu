#include "hip/hip_runtime.h"
#include "head.h"


extern double *d_t;
extern double *d_dt;
extern double *d_it;
extern double *d_V;
extern double *d_dVdt;
extern double *d_Vnew;
extern double *belta;
extern double *y_temp;
extern double *f;
extern int *d_kk0, *d_kk;

__global__ void boundary(double *d_V){
	int k = blockDim.x * blockIdx.x + threadIdx.x;//这是global index

	if(k<nx){

	    d_V[(k+1)*(nx+2)] = d_V[(k+1)*(nx+2)+1];
        d_V[(k+1)*(nx+2)+(nx+1)] = d_V[(k+1)*(nx+2)+nx];
        d_V[k+1] = d_V[k+1+(nx+2)];
        d_V[(ny+1)*(nx+2)+k+1] = d_V[ny*(nx+2)+k+1];

	}
}

void gpu_Boun(){
	int bpg;
	//tpb = 256;
	bpg = (nx+tpb-1)/tpb;  // 边界条件只需要1列线程,算四条语句
	boundary<<<bpg, tpb>>>(d_V);
	hipDeviceSynchronize();
}

//*********** step 1,  --- sweep in x-direction, Thomas algorithm used to solve tridiagonal linear equations ADI method*******
__global__ void step_1(double *d_V ,double *belta ,double *y_temp ,double *f){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<nx*ny){
		int j = (int)(k/nx);//j =0,1,2,...,n-1。 
		int id = k+(nx+2)+1+(2*j);
		int i = (int)(k/nx);//i =0,1,2,...,n-1。此处i会不会有问题？

		//double belta[nx + 1];
		double eps = D / (dx*dx);
		double eta= eps*dt_max;//这里的时间步长为什么一直是dt_max??是否应该跟着v变而变？
		double b = 1+eta;
		double c = -eta/2; 
		double c_1 = -eta;
		double a = c;
		double a_n = c_1;
		//double f[nx + 1][ny + 1];
		//for (int j = 1; j < ny + 1; j++){
			//for (int i = 1; i < nx + 1; i++){
				if (j==0){
					//f[i][j] = V[i][j]  + (eta/2)*(V[i][j] - 2 * V[i][j] + V[i][j + 1]);
					f[j*nx+i] = d_V[id]  + (eta/2)*(d_V[id] - 2 * d_V[id] + d_V[id+(nx+2)]);
				}else if (j==ny-1){
					//f[i][j] = V[i][j] + (eta/2)*(V[i][j - 1] - 2 * V[i][j] + V[i][j]);
					f[j*nx+i] = d_V[id] + (eta/2)*(d_V[id-(nx+2)] - 2 * d_V[id] + d_V[id]);
				}else{
					//f[i][j] = V[i][j] + (eta/2)*(V[i][j - 1] - 2 * V[i][j] + V[i][j + 1]);
					f[j*nx+i] = d_V[id] + (eta/2)*(d_V[id-(nx+2)] - 2 * d_V[id] + d_V[id+(nx+2)]);
				}
			//}
		//}
		//double y_temp[nx + 1];
		//for (int j = 1; j < ny + 1; j++){
			belta[0] = c_1 / b;
			//y_temp[1] = f[1][j] / b;
			y_temp[0] = f[0+nx*j] / b;    //j =0,1,2,...,n-1。                  
			//for (int i = 2; i < nx; i++){ //i = 2,3,...,n-1
			if(i>0&&i<nx-1){
				//belta[i] = c/(b-a*belta[i-1]);
				belta[i] = c/(b-a*belta[i-1]);
				//y_temp[i] = (f[i][j] - a*y_temp[i - 1]) / (b-a*belta[i-1]);
				y_temp[i] = (f[j*nx+i] - a*y_temp[i - 1]) / (b-a*belta[i-1]);
			}
			//}
			//y_temp[nx] = (f[nx][j] - a_n*y_temp[nx - 1]) / (b - a_n*belta[nx - 1]);
			y_temp[nx-1] = (f[(nx-1)+nx*j] - a_n*y_temp[(nx - 1)-1]) / (b - a_n*belta[(nx - 1)-1]);
			//V[nx][j] = y_temp[nx];
			d_V[(nx+2)+1+(2*j)+(nx-1)+nx*j] = y_temp[nx-1];
			//for (i = nx-1; i >=1; i--){
				//V[i][j] = y_temp[i] - belta[i] * V[i+1][j];
				if(id!=((nx+2)+1+(2*j)+(nx-1)+nx*j))d_V[id] = y_temp[i] - belta[i] * d_V[id+1];//此处k/nx与i有区别吗？
			//}    //int id = k+(nx+2)+1+(2*j);
		//}
	}
}
		//*********** step 1 *******		
void gpuStep_1(){
	int bpg;
	//tpb = 256;	
    bpg = (nx*ny+tpb-1)/tpb;
	step_1<<<bpg, tpb>>>(d_V ,belta ,y_temp ,f);
	hipDeviceSynchronize();
}	
			//*********** part of the step 2 *******	
		//dt = dt_max;//这里的时间步长为什么一直是dt_max??是否应该跟着v变而变？
	/*	for (i = 1; i < nx + 1; i++){
			for (j = 1; j < ny + 1; j++){
				it[i][j] = get_it(i, j);
				dVdt[i][j] = -it[i][j];
			}
		}*/
__global__ void comp_dVdt(double *d_dVdt  ,double *d_it){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if(k<nx*ny){
	//int j = (int)(k/nx);
	//int id = k+(nx+2)+1+(2*j);//这是什么index？
	//		dVdt[i][j] = -it[i][j];
	d_dVdt[k] = -d_it[k];
	}
}	
void gpu_dVdt(){
	int bpg;
	//tpb = 256;	
    bpg = (nx*ny+tpb-1)/tpb;
	comp_dVdt<<<bpg, tpb>>>(d_dVdt, d_it);
	hipDeviceSynchronize();
}			
		//*****stimulation with a plane waves****
		/*if (ncount >= 1 && ncount <= stimtime) { //stimulus is hold with 0.6 ms
			for (i = 1; i < nx + 1; i++){
				for (j = 1; j <= 5; j++){
					dVdt[i][j] = dVdt[i][j] + (-st);
				}
			}
		}*/
__global__ void plane_waves(double *d_dVdt){
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if(k<ny*5){
	int i, j;
	i = (int)(k/nx);
	j = k-i*nx;

	d_dVdt[j*ny+i] = d_dVdt[j*ny+i] + (-st);

	}
}

void gpu_stimu(){
	int bpg;
        //int tpb;
        //tpb = 256;
    bpg = (ny*5+tpb-1)/tpb;    // 因为刺激5列，所以开5列线程就够了
	plane_waves<<<bpg, tpb>>>(d_dVdt);
	hipDeviceSynchronize();
}	
	
__global__ void adaptiveT(double *d_dVdt  ,double *d_dt,int *d_kk,int *d_kk0){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if(k<nx*ny){
		
		//int d_k0, d_k;
		//for (i = 1; i < nx + 1; i++){
			//for (j = 1; j < ny + 1; j++){
				// adaptive time step
				if (d_dVdt[k] > 0){
					d_kk0[0] = 5;
				}else{
					d_kk0[0] = 1;
				}
				d_kk[0] = d_kk0[0] + (int)(fabs(d_dVdt[k]) + 0.5); //round the value此处(dVdt[k])+0.5是为了四舍五入
				if (d_kk[0] >(int)(dt_max / dt_min)){
					d_kk[0] = (int)(dt_max / dt_min);
				}
				d_dt[0] = dt_max / d_kk[0];
			//}
		//}
	}
}	
void gpu_adaptiveT(){
	int bpg;
        //int tpb;
        //tpb = 256;
    bpg = (nx*ny+tpb-1)/tpb;
	adaptiveT<<<bpg, tpb>>>(d_dVdt, d_dt,d_kk,d_kk0);
	hipDeviceSynchronize();
}	
__global__ void Euler(double *d_V, double *d_dVdt, double *d_Vnew, double *d_dt, double *d_t){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if(k<nx*ny){

	int j = (int)(k/nx);
	d_Vnew[k] = d_V[k+nx+2+1+2*j] + d_dt[0]*d_dVdt[k];
    d_V[k+nx+2+1+2*j] = d_Vnew[k];

	}

	if(k==0){

	d_t[0] = d_t[0] + d_dt[0];//此处必须加上数值类型，因为d_dt是一个指针，与d_t[0]是不同类型，否则报错。

	}
	
}

void Forward_Euler(){
	int bpg;
        //int tpb;
        //tpb = 256;
        bpg = (nx*ny+tpb-1)/tpb;
	Euler<<<bpg, tpb>>>(d_V, d_dVdt, d_Vnew, d_dt, d_t);
	hipDeviceSynchronize();
}
		//*********** part of the step 2 *******

		//*********** step 3, sweep in y-direction, Thomas algorithm used to solve tridiagonal linear equations ADI method*******
__global__ void step_3(double *d_V ,double *belta ,double *y_temp ,double *f){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int j = (int)(k/nx);//j =0,1,2,...,n-1。 
	int id = k+(nx+2)+1+(2*j);
	int i = (int)(k/nx);//i =0,1,2,...,n-1。
	if(k<nx*ny){
		
		
		//belta[nx + 1];
		double eta = dt_max*D / (dy*dy);//这里的时间步长为什么一直是dt_max??
		double b = 1+eta;
		double c = -eta/2;
		double c_1 = -eta;
		double a = c;
		double a_n = c_1;
		//for (i = 1; i < nx + 1; i++){
			//for (j = 1; j < ny + 1; j++){
				if (i==1){
					//f[i][j] = V[i][j] + (eta / 2)*(V[i][j] - 2 * V[i][j] + V[i + 1][j]);
					f[j*nx+i] = d_V[id]  + (eta/2)*(d_V[id] - 2 * d_V[id] + d_V[id+1]);
				}else if (i==nx){
					//f[i][j] = V[i][j] + (eta / 2)*(V[i - 1][j] - 2 * V[i][j] + V[i][j]);
					f[j*nx+i] = d_V[id]  + (eta/2)*(d_V[id-1] - 2 * d_V[id] + d_V[id]);
				}else{
					//f[i][j] = V[i][j] + (eta / 2)*(V[i - 1][j] - 2 * V[i][j] + V[i + 1][j]);
					f[j*nx+i] = d_V[id]  + (eta/2)*(d_V[id-1] - 2 * d_V[id] + d_V[id+1]);
				}
			//}
		//}

		//y_temp[nx + 1] ;
		//for (i = 1; i < nx + 1; i++){
			//belta[1] = c_1 / b;
			belta[0] = c_1 / b;
			//y_temp[1] = f[i][1] / b;
			y_temp[0] = f[i+nx*0] / b; 
			//for (j = 2; j < ny; j++){ 
			if(j>0&&j<nx-1){
				//belta[j] = c / (b - a*belta[j - 1]);
				belta[j] = c/(b-a*belta[j-1]);
				//y_temp[j] = (f[i][j] - a*y_temp[j - 1]) / (b - a*belta[j - 1]);
				y_temp[j] = (f[j*nx+i] - a*y_temp[j - 1]) / (b-a*belta[j-1]);
			}
			//y_temp[ny] = (f[i][ny] - a_n*y_temp[ny - 1]) / (b - a_n*belta[ny - 1]);
			y_temp[ny-1] = (f[i+nx*ny] - a_n*y_temp[(ny - 1)-1]) / (b - a_n*belta[(ny - 1)-1]);
			//V[i][ny] = y_temp[ny];
			d_V[(nx+2)+1+(2*j)+i+nx*(ny-1)] = y_temp[ny-1];
			//for (j = ny - 1; j >= 1; j--){
				//V[i][j] = y_temp[j] - belta[j] * V[i][j + 1];
				if(id!=((nx+2)+1+(2*j)+i+nx*(ny-1)))d_V[id] = y_temp[j] - belta[j] * d_V[id+(nx+2)];
			//}
		}
}
void gpuStep_3(){
	int bpg;
	//tpb = 256;	
    bpg = (nx*ny+tpb-1)/tpb;
	step_3<<<bpg, tpb>>>(d_V ,belta ,y_temp ,f);
	hipDeviceSynchronize();
}	
	
		//*********** step 3 *******

		//t = t + dt_max;