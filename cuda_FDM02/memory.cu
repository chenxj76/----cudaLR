#include "head.h"

double *h_t;
double *d_t;
double *h_dt;
double *d_dt;

double *h_V;
double *d_V;
double *h_dVdt;
double *d_dVdt;
double *d_dV2;
double *h_Vnew;
double *d_Vnew;

double *h_m;
double *h_m0;
double *d_m;
double *d_m0;
double *h_h;
double *h_h0;
double *d_h;
double *d_h0;
double *h_jj;
double *h_jj0;
double *d_jj;
double *d_jj0;
double *h_d;
double *h_d0;
double *d_d;
double *d_d0;
double *h_f;
double *h_f0;
double *d_f;
double *d_f0;
double *h_X;
double *h_X0;
double *d_X;
double *d_X0;
double *h_cai;
double *d_cai;

double *isi;
double *esi;
double *ina;
double *ik;
double *ik1;
double *ikp;
double *ib;
double *h_it;
double *d_it;

double *f;//double f[nx + 1][ny + 1];
double *belta;//double belta[nx + 1];
double *y_temp;//double y_temp[nx + 1];

int *d_kk0, *d_kk, *h_kk, *h_kk0;

void Allocate(){
	hipError_t Error;
	size_t size = nx*ny*sizeof(double);
	size_t size2 = nx*ny*sizeof(int);

	h_t = (double*)malloc(sizeof(double));
	Error = hipMalloc((void**)&d_t, sizeof(double));
	printf("CUDA error = %s\n",hipGetErrorString(Error));	
	h_dt = (double*)malloc(size);
		hipMalloc((void**)&d_dt, size);
			
	h_V = (double*)malloc((nx+2)*(ny+2)*sizeof(double));
		hipMalloc((void**)&d_V, (nx+2)*(ny+2)*sizeof(double));
	h_dVdt = (double*)malloc(size);
		hipMalloc((void**)&d_dVdt, size);
	h_Vnew = (double*)malloc(size);
		Error = hipMalloc((void**)&d_Vnew, size);
		printf("CUDA d_Vnew error = %s\n",hipGetErrorString(Error));
		hipMalloc((void**)&d_dV2, size);		
		
	h_m = (double*)malloc(size);
	h_m0= (double*)malloc(size);
		hipMalloc((void**)&d_m, size);
		hipMalloc((void**)&d_m0, size);
	h_h = (double*)malloc(size);
	h_h0 = (double*)malloc(size);
		hipMalloc((void**)&d_h, size);
		hipMalloc((void**)&d_h0, size);
	h_jj = (double*)malloc(size);
	h_jj0 = (double*)malloc(size);
		hipMalloc((void**)&d_jj, size);
		hipMalloc((void**)&d_jj0, size);
	h_d = (double*)malloc(size);
	h_d0 = (double*)malloc(size);
        hipMalloc((void**)&d_d, size);
		hipMalloc((void**)&d_d0, size);
	h_f = (double*)malloc(size);
	h_f0 = (double*)malloc(size);
        hipMalloc((void**)&d_f, size);
		hipMalloc((void**)&d_f0, size);
	h_X = (double*)malloc(size);
	h_X0 = (double*)malloc(size);
        hipMalloc((void**)&d_X, size);
		hipMalloc((void**)&d_X0, size);	
	h_cai = (double*)malloc(size);	
        hipMalloc((void**)&d_cai, size);
		
		hipMalloc((void**)&isi, size);
		hipMalloc((void**)&esi, size);
		hipMalloc((void**)&ina, size);
		hipMalloc((void**)&ik, size);
		hipMalloc((void**)&ik1, size);
		hipMalloc((void**)&ikp, size);
		hipMalloc((void**)&ib, size);
	h_it = (double*)malloc(size);
		hipMalloc((void**)&d_it, size);
	
	hipMalloc((void**)&f, size);
	hipMalloc((void**)&belta, nx*sizeof(double));
	hipMalloc((void**)&y_temp, nx*sizeof(double));
	
	h_kk = (int*)malloc(size2);
	h_kk0 = (int*)malloc(size2);
		hipMalloc((void**)&d_kk, size2);
		hipMalloc((void**)&d_kk0, size2);		
}

void free(){

	free(h_t);free(h_dt);hipFree(d_dt);hipFree(d_t);
	
	free(h_V);hipFree(d_V);free(h_dVdt);hipFree(d_dVdt);//free(h_Vnew);hipFree(d_Vnew);
	free(h_Vnew);hipFree(d_Vnew);hipFree(d_dV2);
	
	free(h_m);free(h_h);free(h_jj);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);
	free(h_m0);free(h_h0);free(h_jj0);
	hipFree(d_m0);hipFree(d_h0);hipFree(d_jj0);
	free(h_d);free(h_f);free(h_X);
	hipFree(d_d);hipFree(d_f);hipFree(d_X);
	free(h_d0);free(h_f0);free(h_X0);
	hipFree(d_d0);hipFree(d_f0);hipFree(d_X0);
	free(h_cai);hipFree(d_cai);
	
	hipFree(isi);hipFree(esi);hipFree(ina);
	hipFree(ik);hipFree(ik1);hipFree(ikp);hipFree(ib);
	free(h_it);hipFree(d_it);
	
	hipFree(f);hipFree(belta);hipFree(y_temp);
	
	free(h_kk0);free(h_kk);hipFree(d_kk);hipFree(d_kk0);			
}



void Manage_Comms(int phase){
        hipError_t Error;
		size_t size = nx*ny*sizeof(double);
		size_t size2 = nx*ny*sizeof(int);
if (phase==1){
        Error = hipMemcpy(d_dt, h_dt, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_dt->d_dt) = %s\n",hipGetErrorString(Error));
		Error = hipMemcpy(d_kk, h_kk, size2, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_kk->d_kk) = %s\n",hipGetErrorString(Error));
		Error = hipMemcpy(d_kk0, h_kk0, size2, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_kk0->d_kk0) = %s\n",hipGetErrorString(Error));
}

if (phase==2){   
        Error = hipMemcpy(h_dt, d_dt, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_dt->h_dt) = %s\n",hipGetErrorString(Error));
}

if (phase==3){
        Error = hipMemcpy(h_kk, d_kk, size2, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Kk->h_Kk) = %s\n",hipGetErrorString(Error));
}
if (phase==4){
        Error = hipMemcpy(h_Vnew,d_Vnew,size,hipMemcpyDeviceToHost);
       if (Error != hipSuccess)printf("CUDA error(copy d_Vnew->h_Vnew) = %s\n",hipGetErrorString(Error));
}
if (phase==5){
		hipError_t Error;     
        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));      
}
}



void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

	Error = hipMemcpy(d_t, h_t, sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}

void Send_V(){
        hipError_t Error;
        size_t size;
        //size = nx*ny*sizeof(double);
		//Error = hipMemcpy(h_V, d_dV2, size, hipMemcpyDeviceToHost);
		size = (nx+2)*(ny+2)*sizeof(double);
        Error = hipMemcpy(h_V, d_V, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));
}

void Send_dVdt(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);
		Error = hipMemcpy(h_dVdt, d_dVdt, size, hipMemcpyDeviceToHost);
		//size = (nx+2)*(ny+2)*sizeof(double);
        //Error = hipMemcpy(h_V, d_V, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_dVdt->d_dVdt) = %s\n",hipGetErrorString(Error));
}

