#include "head.h"

double *h_t;
double *d_t;
double *h_dt;
double *d_dt;
double *h_V;
double *d_V;
double *d_dVdt;
double *h_Vnew;
double *d_Vnew;
double *d_it;

double *h_m;
double *d_m;
double *d_m0;
double *h_h;
double *d_h;
double *d_h0;
double *h_jj;
double *d_jj;
double *d_jj0;
double *h_d;
double *d_d;
double *d_d0;
double *h_f;
double *d_f;
double *d_f0;
double *h_X;
double *d_X;
double *d_X0;
double *h_cai;
double *d_cai;

double *h_it;

double *f;//double f[nx + 1][ny + 1];
double *belta;//double belta[nx + 1];
double *y_temp;//double y_temp[nx + 1];

int *d_kk0, *d_kk, *h_kk;

void Allocate(){
	hipError_t Error;
	size_t size = nx*ny*sizeof(double);

	h_t = (double*)malloc(sizeof(double));
	Error = hipMalloc((void**)&d_t, sizeof(double));
	printf("CUDA error = %s\n",hipGetErrorString(Error));	
	h_dt = (double*)malloc(sizeof(double));
		hipMalloc((void**)&d_dt, sizeof(double));
	h_kk = (int*)malloc(sizeof(int));
		hipMalloc((void**)&d_kk, sizeof(int));
	h_V = (double*)malloc((nx+2)*(ny+2)*sizeof(double));
		hipMalloc((void**)&d_V, (nx+2)*(ny+2)*sizeof(double));
		hipMalloc((void**)&d_dVdt, size);
	h_Vnew = (double*)malloc(size);
		hipMalloc((void**)&d_Vnew, size);

		hipMalloc((void**)&d_it, size);

	h_m = (double*)malloc(size);
		hipMalloc((void**)&d_m, size);
		hipMalloc((void**)&d_m0, size);
	h_h = (double*)malloc(size);
		hipMalloc((void**)&d_h, size);
		hipMalloc((void**)&d_h0, size);
	h_jj = (double*)malloc(size);
		hipMalloc((void**)&d_jj, size);
		hipMalloc((void**)&d_jj0, size);
	h_d = (double*)malloc(size);
        hipMalloc((void**)&d_d, size);
		hipMalloc((void**)&d_d0, size);
	h_f = (double*)malloc(size);
        hipMalloc((void**)&d_f, size);
		hipMalloc((void**)&d_f0, size);
	h_X = (double*)malloc(size);
        hipMalloc((void**)&d_X, size);
		hipMalloc((void**)&d_X0, size);
	h_cai = (double*)malloc(size);
        hipMalloc((void**)&d_cai, size);

	h_it = (double*)malloc(size);
	
	hipMalloc((void**)&f, size);
	hipMalloc((void**)&belta, nx*sizeof(double));
	hipMalloc((void**)&y_temp, nx*sizeof(double));
}

void free(){

	free(h_t);free(h_V);free(h_m);free(h_h);
	free(h_jj);free(h_d);free(h_f);free(h_X);free(h_cai);
	free(h_Vnew);
	free(h_it);
	free(h_dt);hipFree(d_dt);
	free(h_kk);hipFree(d_kk);
	hipFree(d_t);hipFree(d_V);hipFree(d_dVdt);hipFree(d_Vnew);hipFree(d_it);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);hipFree(d_d);
	hipFree(d_m0);hipFree(d_h0);hipFree(d_jj0);hipFree(d_d0);
	hipFree(d_f);hipFree(d_X);hipFree(d_f0);hipFree(d_X0);hipFree(d_cai);
	hipFree(f);hipFree(belta);hipFree(y_temp);
}

void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

	Error = hipMemcpy(d_t, h_t, sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
}
/*
void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}
*/
void Send_V(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_Vnew,d_Vnew,size,hipMemcpyDeviceToHost);
        if (Error != hipSuccess)printf("CUDA error(copy d_Vnew->h_Vnew) = %s\n",hipGetErrorString(Error));
}
void Send2deviceT(){
        hipError_t Error;
        size_t size;
        size = sizeof(double);

        Error = hipMemcpy(d_dt, h_dt, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_dt->d_dt) = %s\n",hipGetErrorString(Error));
}
void Send2hostT(){
        hipError_t Error;
        size_t size;
        size = sizeof(double);

        Error = hipMemcpy(h_dt, d_dt, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_dt->h_dt) = %s\n",hipGetErrorString(Error));
}
void Send2hostK(){
        hipError_t Error;
        size_t size;
        size = sizeof(int);

        Error = hipMemcpy(h_kk, d_kk, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Kk->h_Kk) = %s\n",hipGetErrorString(Error));
}

