#include "hip/hip_runtime.h"
/*
void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}
*/
-arch=sm_61
__global__ void comp_ina(double *d_V, double *d_m, double *d_h, double *d_jj, 
		double *d_m0, double *d_h0, double *d_jj0, double *d_dt, double *d_it);
__global__ void comp_ical(double *d_V, double *d_d, double *d_f, double *d_d0, 
						double *d_f0, double *d_cai, double *d_dt, double *d_it);
__global__ void comp_ik(double *d_V, double *d_X, double *d_X0, double *d_dt,double *d_it);
__global__ void comp_ik1(double *d_V, double *d_it);
__global__ void comp_ikp(double *d_V, double *d_it);
__global__ void comp_ib(double *d_V, double *d_it);
__global__ void new_gate(double *d_m, double *d_h, double *d_jj,
						double *d_m0, double *d_h0, double *d_jj0,
						double *d_d, double *d_f, double *d_d0, double *d_f0, 
						double *d_X,double *d_X0);