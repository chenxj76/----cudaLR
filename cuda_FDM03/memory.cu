#include "head.h"

double *h_t,*d_t;
double *h_V,*d_V,*d_dV2,*h_Vnew,*d_Vnew;
double *h_m,*d_m,*h_h,*d_h,*h_jj,*d_jj;
double *h_d,*d_d,*h_f,*d_f;
double *h_X,*d_X,*h_cai,*d_cai,*h_it,*d_it;
//double *d_esi,*d_isi,*h_esi,*h_isi;

void Allocate(){
	hipError_t Error;
	size_t size = nx*ny*sizeof(double);

	h_t = (double*)malloc(sizeof(double));
	Error = hipMalloc((void**)&d_t, sizeof(double));
	printf("CUDA error = %s\n",hipGetErrorString(Error));

	h_V = (double*)malloc((nx+2)*(ny+2)*sizeof(double));
	hipMalloc((void**)&d_V, (nx+2)*(ny+2)*sizeof(double));
	hipMalloc((void**)&d_dV2, size);
	h_Vnew = (double*)malloc(size);
	hipMalloc((void**)&d_Vnew, size);

	

	h_m = (double*)malloc(size);
	hipMalloc((void**)&d_m, size);
	h_h = (double*)malloc(size);
        hipMalloc((void**)&d_h, size);
	h_jj = (double*)malloc(size);
        hipMalloc((void**)&d_jj, size);
	h_d = (double*)malloc(size);
        hipMalloc((void**)&d_d, size);
	h_f = (double*)malloc(size);
        hipMalloc((void**)&d_f, size);
	h_X = (double*)malloc(size);
        hipMalloc((void**)&d_X, size);
	h_cai = (double*)malloc(size);
        hipMalloc((void**)&d_cai, size);
		/*
		h_esi= (double*)malloc(size);
		hipMalloc((void**)&d_esi, size);
		h_isi= (double*)malloc(size);
		hipMalloc((void**)&d_isi, size);
*/
	h_it = (double*)malloc(size);
	hipMalloc((void**)&d_it, size);
}

void free(){

	free(h_t);free(h_V);free(h_m);free(h_h);
	free(h_jj);free(h_d);free(h_f);free(h_X);free(h_cai);
	free(h_Vnew);
	free(h_it);
	//free(h_esi);free(h_isi);

	hipFree(d_t);hipFree(d_V);hipFree(d_dV2);hipFree(d_Vnew);hipFree(d_it);
	hipFree(d_m);hipFree(d_h);hipFree(d_jj);hipFree(d_d);
	hipFree(d_f);hipFree(d_X);hipFree(d_cai);
	//hipFree(d_esi);hipFree(d_isi);
}

void Send_to_Device(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

	Error = hipMemcpy(d_t, h_t, sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_t->d_t) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(d_V, h_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_V->d_V) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_m, h_m, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_m->d_m) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_h, h_h, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_h->d_h) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_jj, h_jj, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_jj->d_jj) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_d->d_d) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_f, h_f, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_f->d_f) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_X->d_X) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_cai, h_cai, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_cai->d_cai) = %s\n",hipGetErrorString(Error));
	/*
	Error = hipMemcpy(d_esi, h_esi, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_esi->d_esi) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_isi, h_isi, size, hipMemcpyHostToDevice);
        if (Error != hipSuccess)
        printf("CUDA error(copy h_isi->d_isi) = %s\n",hipGetErrorString(Error));
	*/
}

void Send_to_Host(){
	hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_V, d_V, (nx+2)*(ny+2)*sizeof(double), hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_V->h_V) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_m, d_m, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_m->h_m) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_h, d_h, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_h->h_h) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_jj, d_jj, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_jj->h_jj) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_d, d_d, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_d->h_d) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_f, d_f, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_f->h_f) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_X, d_X, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_X->h_X) = %s\n",hipGetErrorString(Error));
        Error = hipMemcpy(h_cai, d_cai, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
	printf("CUDA error(copy d_cai->h_cai) = %s\n",hipGetErrorString(Error));
}

void Send_V(){
        hipError_t Error;
        size_t size;
        size = nx*ny*sizeof(double);

        Error = hipMemcpy(h_Vnew, d_Vnew, size, hipMemcpyDeviceToHost);
        if (Error != hipSuccess)
        printf("CUDA error(copy d_Vnew->Vnew) = %s\n",hipGetErrorString(Error));
}

void Save_Result(){

        FILE *pFile;
        int i,j;
        int index;
        //int n;
        //n = nx;
        pFile = fopen("V.txt","w+");
        // Save the matrix V
        for (i = 0; i < ny; i++) {
                for (j = 0; j < nx; j++) {
                        index = i*nx + j;
                        fprintf(pFile, "%g", h_Vnew[index]);
                        if (j == (nx-1)) {
                                fprintf(pFile, "\n");
                        }else{
                                fprintf(pFile, "\t");
                        }
                }
        }
        fclose(pFile);

}

